
#include <hip/hip_runtime.h>
__global__ void csrMatrixVectorMultKernel(const unsigned* csrRows, const unsigned* csrCols, const float*csrVals, const float* v1, float* rv,const unsigned rows)
{
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rows) return;

    const unsigned rowStart = csrRows[row]; 
    const unsigned rowEnd   = csrRows[row + 1];

    rv[row] = 0;

    for(unsigned i = rowStart ; i < rowEnd ; i++ )
    {
        rv[row] += csrVals[i] * v1[csrCols[i]]; //TODO: use an acc to limit global access memory
    }
}