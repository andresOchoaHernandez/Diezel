
#include <hip/hip_runtime.h>
__global__ void vectorDifKernel(const int* v1, const int* v2, int* rv, const unsigned size)
{
    const unsigned globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(globalIndex >= size) return;

    rv[globalIndex] = v1[globalIndex] - v2[globalIndex];
}

__global__ void matrixVectorMultKernel(const int* matrix, const int* v1, int* rv, const unsigned rows, const unsigned cols)
{
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= rows) return;

    int acc = 0;

    for(unsigned i = 0u; i < cols; i++ )
    {
        acc+= matrix[row * cols + i] * v1[i];
    }    

    rv[row] = acc;
}

__global__ void csrMatrixVectorMultKernel(const int* csrRows, const int* csrCols, const int*csrVals, const int* v1, int* rv,const unsigned rows)
{
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rows) return;

    const unsigned rowStart = csrRows[row]; 
    const unsigned rowEnd   = csrRows[row + 1];

    rv[row] = 0;

    for(unsigned i = rowStart ; i < rowEnd ; i++ )
    {
        rv[row] += csrVals[i] * v1[csrCols[i]];
    }
}