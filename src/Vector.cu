#include <random>
#include <chrono>
#include <iostream>

#include "LinearAlgebra.hpp"
#include "VectorKernels.cu"

namespace LinearAlgebra
{
    Vector::Vector(unsigned len):_len{len},_vec{new double[_len]}{}
    Vector::Vector(const Vector& vector):_len{vector._len},_vec{new double[_len]}
    {
        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++)
        {
            _vec[i] = vector._vec[i];
        }
    }
    Vector::Vector(Vector&& v)
    {
        _len = v._len;
        _vec = v._vec;
        v._len = 0u;
        v._vec = nullptr;
    }
    Vector::~Vector(){delete[] _vec;}


    Vector Vector::operator+(const Vector& other)const
    {
        if( _len != other.len()) throw std::runtime_error{"Vectors dimensions don't match"};

        Vector result{_len};
        
        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] + other[i];
        }

        return result;
    }

    Vector Vector::operator-(const Vector& other)const
    {
        if( _len != other.len()) throw std::runtime_error{"Vectors dimensions don't match"};
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i]- other[i];
        }

        return result;
    }

    Vector Vector::operator*(const Vector& other)const
    {
        if( _len != other.len()) throw std::runtime_error{"Vectors dimensions don't match"};
        
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i]*other[i];
        }

        return result;
    }

    Vector Vector::operator/(const Vector& other)const
    {
        if( _len != other.len()) throw std::runtime_error{"Vectors dimensions don't match"};
        
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] / other[i];
        }

        return result;
    }

    Vector Vector::operator+(const double constant)const
    {   
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] + constant;
        }

        return result;
    }

    Vector Vector::operator-(const double constant)const
    {
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] - constant;
        }

        return result;
    }

    Vector Vector::operator*(const double constant)const
    {
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] * constant;
        }

        return result;
    }

    Vector Vector::operator/(const double constant)const
    {
        Vector result{_len};

        #pragma omp parallel for
        for(unsigned i = 0u ; i < _len ; i++ )
        {
            result[i] = _vec[i] / constant;
        }

        return result;
    }


    Vector Vector::gpu_diff(const Vector& v2)const
    {
        if( _len != v2.len()) throw std::runtime_error{"Vectors dimensions don't match"};

        Vector rv{_len};

        double *v1_device;double *v2_device;double *rv_device;

        hipMalloc(&v1_device,sizeof(double)*_len);
        hipMalloc(&v2_device,sizeof(double)*v2.len());
        hipMalloc(&rv_device,sizeof(double)*rv.len());

        hipMemcpy(v1_device,_vec,sizeof(double)*_len,hipMemcpyHostToDevice);
        hipMemcpy(v2_device,&v2[0u],sizeof(double)*v2.len(),hipMemcpyHostToDevice);

        const unsigned threadsPerBlock = 1024u;
        const unsigned numberOfBlocks = _len < threadsPerBlock? 1u: (_len % threadsPerBlock == 0u? _len/threadsPerBlock:_len/threadsPerBlock +1u);
        dim3 dimGrid(numberOfBlocks,1,1);
        dim3 dimBlock(threadsPerBlock,1,1);
        
        vectorDifKernel<<<dimGrid,dimBlock>>>(v1_device,v2_device,rv_device,_len);
        hipDeviceSynchronize();

        hipMemcpy(&rv[0u],rv_device,sizeof(double)*rv.len(),hipMemcpyDeviceToHost);

        hipFree(v1_device);
        hipFree(v2_device);
        hipFree(rv_device);

        hipDeviceReset();

        return rv;
    }

    Vector Vector::gpu_sum(const Vector& v2)const
    {
        //TODO:
        return v2;
    }

    void Vector::randomInit(double a, double b)
    {
        std::random_device dev;
        std::mt19937 rng(dev());
        std::uniform_real_distribution<double> dist(a,b);

        #pragma omp parallel for
        for (unsigned i = 0u ; i < _len ; i++ )
            _vec[i] = dist(rng);
    }

    void Vector::valInit(double val)
    {
        #pragma omp parallel for
        for (unsigned i = 0u ; i < _len ; i++ )
            _vec[i] = val;
    }

    unsigned Vector::len()const{ return _len; }
    double* Vector::getVec(){ return _vec; }

    double& Vector::operator [](unsigned i){return _vec[i];}
    const double& Vector::operator [](unsigned i)const{return _vec[i];}

    bool Vector::operator==(const Vector& other) const
    {
        if(_len != other.len()) return false;

        for(unsigned i = 0u ; i < _len ; i++)
        {
            if(_vec[i] != other[i]) return false;
        }

        return true;
    }

    std::ostream& operator<<(std::ostream& stream, const Vector& operand)
    {
        for(unsigned i = 0u ; i < operand._len ; i++)
            stream << operand[i] << " ";
    
        stream << std::endl;

        return stream;
    }
}