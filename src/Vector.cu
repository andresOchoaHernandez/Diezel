#include <random>
#include <chrono>
#include <iostream>
#include <vector>

#include "LinearAlgebra.hpp"
#include "VectorKernels.cu"

namespace LinearAlgebra
{
    Vector::Vector(unsigned len):_len{len},_vec{new int[_len]}{}
    Vector::Vector(const Vector& vector):_len{vector._len},_vec{new int[_len]}{for(unsigned i = 0u ; i < _len ; i++)_vec[i] = vector._vec[i];}
    Vector::Vector(Vector&& v)
    {
        _len = v._len;
        _vec = v._vec;
        v._len = 0u;
        v._vec = nullptr;
    }
    Vector::~Vector(){delete[] _vec;}

    Vector Vector::gpuVectorDif(const Vector& v2)const
    {
        if( _len != v2.len()) throw std::runtime_error{"Vectors dimensions don't match"};

        Vector rv{_len};

        int *v1_device;int *v2_device;int *rv_device;

        hipMalloc(&v1_device,sizeof(int)*_len);
        hipMalloc(&v2_device,sizeof(int)*v2.len());
        hipMalloc(&rv_device,sizeof(int)*rv.len());

        hipMemcpy(v1_device,_vec,sizeof(int)*_len,hipMemcpyHostToDevice);
        hipMemcpy(v2_device,&v2[0u],sizeof(int)*v2.len(),hipMemcpyHostToDevice);

        const unsigned threadsPerBlock = 1024u;
        const unsigned numberOfBlocks = _len < threadsPerBlock? 1u: (_len % threadsPerBlock == 0u? _len/threadsPerBlock:_len/threadsPerBlock +1u);
        dim3 dimGrid(numberOfBlocks,1,1);
        dim3 dimBlock(threadsPerBlock,1,1);

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        vectorDifKernel<<<dimGrid,dimBlock>>>(v1_device,v2_device,rv_device,_len);
        hipDeviceSynchronize();
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "Cuda kernel for vector diff took: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " ms" << std::endl;

        hipMemcpy(&rv[0u],rv_device,sizeof(int)*rv.len(),hipMemcpyDeviceToHost);

        hipFree(v1_device);
        hipFree(v2_device);
        hipFree(rv_device);

        hipDeviceReset();

        return rv;
    }

    Vector Vector::seqVectorDif(const Vector& v2) const
    {
        if( _len != v2.len()) throw std::runtime_error{"Vectors dimensions don't match"};
        
        Vector rv{_len};

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        for(unsigned i = 0u ; i < rv.len(); i++)
        {
            rv[i] = _vec[i] - v2[i];
        }
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "Sequential vector diff took: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " ms" << std::endl;
        
        return rv;
    }

    void Vector::randomInit(int a, int b)
    {
        std::random_device dev;
        std::mt19937 rng(dev());
        std::uniform_int_distribution<std::mt19937::result_type> dist(a,b);

        for (unsigned i = 0u ; i < _len ; i++ )
            _vec[i] = dist(rng);
    }

    void Vector::valInit(int val)
    {
        for (unsigned i = 0u ; i < _len ; i++ )
            _vec[i] = val;
    }

    unsigned Vector::len()const{ return _len; }
    int* Vector::getVec(){ return _vec; }

    int& Vector::operator [](unsigned i){return _vec[i];}
    const int& Vector::operator [](unsigned i)const{return _vec[i];}

    std::ostream& operator<<(std::ostream& stream, const Vector& operand)
    {
        for(unsigned i = 0u ; i < operand._len ; i++)
            stream << operand[i] << " ";
    
        stream << std::endl;

        return stream;
    }
}