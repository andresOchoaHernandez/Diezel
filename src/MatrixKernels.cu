
#include <hip/hip_runtime.h>
__global__ void matrixVectorMultKernel(const int* matrix, const int* v1, int* rv, const unsigned rows, const unsigned cols)
{
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= rows) return;

    int acc = 0;

    for(unsigned i = 0u; i < cols; i++ )
    {
        acc+= matrix[row * cols + i] * v1[i];
    }    

    rv[row] = acc;
}